#include "hip/hip_runtime.h"
/**
 * @file      nbody.cu
 *
 * @author    Tomáš Matuš \n
 *            Faculty of Information Technology \n
 *            Brno University of Technology \n
 *            xmatus37@fit.vutbr.cz
 *
 * @brief     PCG Assignment 1
 *
 * @version   2024
 *
 * @date      04 October   2023, 09:00 (created) \n
 */

#include <>

#include "nbody.cuh"

/* Constants */
constexpr float G                  = 6.67384e-11f;
constexpr float COLLISION_DISTANCE = 0.01f;

/**
 * CUDA kernel to calculate new particles velocity and position
 * @param pIn  - particles in
 * @param pOut - particles out
 * @param N    - Number of particles
 * @param dt   - Size of the time step
 */
__global__ void calculateVelocity(Particles pIn, Particles pOut, const unsigned N, float dt)
{
  /********************************************************************************************************************/
  /*  TODO: CUDA kernel to calculate new particles velocity and position, use shared memory to minimize memory access */
  /********************************************************************************************************************/
  extern __shared__ float sharedMem[];

  // shared mem vectors
  float* const sharedPosX = sharedMem;
  float* const sharedPosY = sharedMem + blockDim.x;
  float* const sharedPosZ = sharedMem + 2 * blockDim.x;
  float* const sharedVelX = sharedMem + 3 * blockDim.x;
  float* const sharedVelY = sharedMem + 4 * blockDim.x;
  float* const sharedVelZ = sharedMem + 5 * blockDim.x;
  float* const sharedWeight = sharedMem + 6 * blockDim.x;

  const unsigned threadID = threadIdx.x + blockIdx.x * blockDim.x;

  // tmp velocity
  float newGravitationVelX{};
  float newGravitationVelY{};
  float newGravitationVelZ{};
  float newCollisionVelX{};
  float newCollisionVelY{};
  float newCollisionVelZ{};

  // current point
  const bool bound = threadID < N;
  const float posX = (bound) ? pIn.posX[threadID] : 0.0f;
  const float posY = (bound) ? pIn.posY[threadID] : 0.0f;
  const float posZ = (bound) ? pIn.posZ[threadID] : 0.0f;
  const float velX = (bound) ? pIn.velX[threadID] : 0.0f;
  const float velY = (bound) ? pIn.velY[threadID] : 0.0f;
  const float velZ = (bound) ? pIn.velZ[threadID] : 0.0f;
  const float weight = (bound) ? pIn.weight[threadID] : 0.0f;

  unsigned tileCount = ceil((float)N / blockDim.x);
  for (unsigned i = 0; i < tileCount; i++) {
    unsigned threadOffset = i * blockDim.x + threadIdx.x;

    // load data to shared memory
    const bool tileBound = threadOffset < N;
    sharedPosX[threadIdx.x] = (tileBound) ? pIn.posX[threadOffset] : 0.0f;
    sharedPosY[threadIdx.x] = (tileBound) ? pIn.posY[threadOffset] : 0.0f;
    sharedPosZ[threadIdx.x] = (tileBound) ? pIn.posZ[threadOffset] : 0.0f;
    sharedVelX[threadIdx.x] = (tileBound) ? pIn.velX[threadOffset] : 0.0f;
    sharedVelY[threadIdx.x] = (tileBound) ? pIn.velY[threadOffset] : 0.0f;
    sharedVelZ[threadIdx.x] = (tileBound) ? pIn.velZ[threadOffset] : 0.0f;
    sharedWeight[threadIdx.x] = (tileBound) ? pIn.weight[threadOffset] : 0.0f;

    __syncthreads();

    // loop over all points in the tile
    for (unsigned j = 0u; j < blockDim.x; j++) {
      // distance between particles in dimensions
      const float dx = sharedPosX[j] - posX;
      const float dy = sharedPosY[j] - posY;
      const float dz = sharedPosZ[j] - posZ;

      // distance r between particles in 3D
      const float r2 = dx * dx + dy * dy + dz * dz;
      const float r = sqrt(r2);

      // gravity force of the two particles
      const float f = G * weight * sharedWeight[j] / r2 + __FLT_MIN__;

      // SUM(F^(i+1))
      if (r > COLLISION_DISTANCE) {
        newGravitationVelX += dx / r * f;
        newGravitationVelY += dy / r * f;
        newGravitationVelZ += dz / r * f;
      } else {
        const bool isColliding = r > 0.0f;
        newCollisionVelX += (isColliding)
                    ? (((weight * velX - sharedWeight[j] * velX + 2.f * sharedWeight[j] * sharedVelX[j]) / (weight + sharedWeight[j])) - velX)
                    : 0.f;
        newCollisionVelY += (isColliding)
                    ? (((weight * velY - sharedWeight[j] * velY + 2.f * sharedWeight[j] * sharedVelY[j]) / (weight + sharedWeight[j])) - velY)
                    : 0.f;
        newCollisionVelZ += (isColliding)
                    ? (((weight * velZ - sharedWeight[j] * velZ + 2.f * sharedWeight[j] * sharedVelZ[j]) / (weight + sharedWeight[j])) - velZ)
                    : 0.f;
      }

    }

    __syncthreads();
  }

  // Final results from the first kernel in step0
  if (bound) {
    newGravitationVelX *= dt / weight;
    newGravitationVelY *= dt / weight;
    newGravitationVelZ *= dt / weight;

    const float nextStepVelX = velX + newGravitationVelX + newCollisionVelX;
    const float nextStepVelY = velY + newGravitationVelY + newCollisionVelY;
    const float nextStepVelZ = velZ + newGravitationVelZ + newCollisionVelZ;

    pOut.posX[threadID] = posX + nextStepVelX * dt;
    pOut.posY[threadID] = posY + nextStepVelY * dt;
    pOut.posZ[threadID] = posZ + nextStepVelZ * dt;

    pOut.velX[threadID] = nextStepVelX;
    pOut.velY[threadID] = nextStepVelY;
    pOut.velZ[threadID] = nextStepVelZ;
  }
}// end of calculate_gravitation_velocity
//----------------------------------------------------------------------------------------------------------------------

/**
 * CUDA kernel to calculate particles center of mass
 * @param p    - particles
 * @param com  - pointer to a center of mass
 * @param lock - pointer to a user-implemented lock
 * @param N    - Number of particles
 */
__global__ void centerOfMass(Particles p, float4* com, int* lock, const unsigned N)
{

}// end of centerOfMass
//----------------------------------------------------------------------------------------------------------------------

/**
 * CPU implementation of the Center of Mass calculation
 * @param particles - All particles in the system
 * @param N         - Number of particles
 */
__host__ float4 centerOfMassRef(MemDesc& memDesc)
{
  float4 com{};

  for (std::size_t i{}; i < memDesc.getDataSize(); i++)
  {
    const float3 pos = {memDesc.getPosX(i), memDesc.getPosY(i), memDesc.getPosZ(i)};
    const float  w   = memDesc.getWeight(i);

    // Calculate the vector on the line connecting current body and most recent position of center-of-mass
    // Calculate weight ratio only if at least one particle isn't massless
    const float4 d = {pos.x - com.x,
                      pos.y - com.y,
                      pos.z - com.z,
                      ((memDesc.getWeight(i) + com.w) > 0.0f)
                        ? ( memDesc.getWeight(i) / (memDesc.getWeight(i) + com.w))
                        : 0.0f};

    // Update position and weight of the center-of-mass according to the weight ration and vector
    com.x += d.x * d.w;
    com.y += d.y * d.w;
    com.z += d.z * d.w;
    com.w += w;
  }

  return com;
}// enf of centerOfMassRef
//----------------------------------------------------------------------------------------------------------------------
